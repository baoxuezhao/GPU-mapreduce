#include "hip/hip_runtime.h"
// MapReduce Inverted Index example using CUDA
// Syntax: invertedindex path-of-data-dir
// (1) assume each host has four processors, each corresponds
//     to a GPU, and read one parts of the files in the local dir
// (2) parse into words separated by whitespace
// (3) count occurrence of each word in all files
// (4) print top 10 words

#include <mpi.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <fstream>
#include <sstream>
#include <cstring>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <dirent.h>
#include "mapreduce.h"
#include "keyvalue.h"

#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/functional.h>

#define CEIL(n,m) ((n)/(m) + (int)((n)%(m) !=0))
#define THREAD_CONF(grid, block, gridBound, blockBound) do {\
	    block.x = blockBound;\
	    grid.x = gridBound; \
		if (grid.x > 65535) {\
		   grid.x = (int)sqrt((double)grid.x);\
		   grid.y = CEIL(gridBound, grid.x); \
		}\
	}while (0)

using namespace MAPREDUCE_NS;
using namespace std;

void mymap(int , KeyValue *, void *);
void myreduce(char *, int, char *, int, int *, KeyValue *, void *);
void mycombine(char *, int, char *, int, int *, KeyValue *, void *);

char inputdir[100];
int  num_file = 1;
int me, nprocs;

//int ncompare(char *, int, char *, int);
//void output(uint64_t, char *, int, char *, int, KeyValue *, void *);

struct Info
{
	int me;
	int nproc;
};

#define START		0x00
#define IN_TAG		0x01
#define IN_ATAG		0x02
#define FOUND_HREF	0x03
#define START_LINK	0x04

struct is_start
{
	__host__ __device__
	bool operator()(const int x)
	{
		return x==1;
	}
};

__global__ void mark(
		char 	*text,
		int 	*d_segmask,
		int 	length)
{
	const int tid_x = blockDim.x * blockIdx.x + threadIdx.x;
	const int tid_y = blockDim.y * blockIdx.y + threadIdx.y;

	const int tid = tid_y * (blockDim.x*gridDim.x) + tid_x;

	if(tid < length)
		d_segmask[tid] = 0;

	if(tid >= length-9)
		return;

	if(text[tid] == '<' &&
			text[tid+1] == 'a' &&
			text[tid+2] == ' ' &&
			text[tid+3] == 'h' &&
			text[tid+4] == 'r' &&
			text[tid+5] == 'e' &&
			text[tid+6] == 'f' &&
			text[tid+7] == '=' &&
			text[tid+8] == '\"')
	{
		d_segmask[tid+9] = 1;
	}
}

__global__ void compute_url_length(
		char 	*d_text,
		int		*d_urloffset,
		int		*d_urllength,
		int 	textlen,
		int		url_num)
{
	const int tid_x = blockDim.x * blockIdx.x + threadIdx.x;
	const int tid_y = blockDim.y * blockIdx.y + threadIdx.y;

	const int tid = tid_y * (blockDim.x*gridDim.x) + tid_x;

	if(tid >= url_num)
		return;

	int start = d_urloffset[tid];

	for(int i=start; i < textlen; i++)
	{
		if(d_text[i] == '\"' || i == textlen-1)
		{
			d_urllength[tid] = i-start;
			d_text[i] = '\0';
			return;
		}
	}
}

/* ---------------------------------------------------------------------- */

//parameters: 
int main(int argc, char **args)
{
	MPI_Init(&argc,&args);

	MPI_Comm_rank(MPI_COMM_WORLD, &me);
	MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
	
	if (argc <= 2)
	{
		if (me == 0) printf("Syntax: invertedindex [input_dir num_file]...\n");
		MPI_Abort(MPI_COMM_WORLD,1);
	}
	
	strcpy(inputdir, args[1]);
	num_file = atoi(args[2]);

	MapReduce *mr = new MapReduce(MPI_COMM_WORLD);
	mr->verbosity = 2;
	mr->timer = 1;

	if(NULL==opendir("/mnt/mrmpi/temp"))
	{
		system("mkdir /mnt/mrmpi");
		system("mkdir /mnt/mrmpi/temp");	
	}
	mr->set_fpath("/mnt/mrmpi/temp");
	mr->memsize = 64;
	//mr->outofcore = 1;

	MPI_Barrier(MPI_COMM_WORLD);
	double tstart = MPI_Wtime();

	//printf("start map %d\n", me);

	int mapitem = mr->map(nprocs, mymap, &me);
	//int nfiles = mr->mapfilecount;

	//mr->compress(mycombine, NULL);

	//printf("start aggregate %d\n", me);

	mr->aggregate(NULL);

	//printf("end aggregate %d\n", me);

	mr->convert();
	//mr->collate(NULL);

	//printf("end convert %d\n", me);


	if(NULL==opendir("/mnt/mrmpi_output"))
	{
		system("mkdir /mnt/mrmpi_output");
	}
	system("rm /mnt/mrmpi_output/InvertedIndex*");

	Info info;
	info.me = me;
	info.nproc = nprocs;

	int reduceitem = mr->reduce(myreduce, &info);

	MPI_Barrier(MPI_COMM_WORLD);
	double tstop = MPI_Wtime();

	/*
	mr->sort_values(&ncompare);

	Count count;
	count.n = 0;
	count.limit = 10;
	count.flag = 0;
	mr->map(mr,output,&count);

	mr->gather(1);
	mr->sort_values(ncompare);

	count.n = 0;
	count.limit = 10;
	count.flag = 1;
	mr->map(mr,output,&count);
	*/

	delete mr;

	//printf("map and reduce item are %d, %d\n", mapitem, reduceitem);

	if (me == 0)
	{
		//printf("%d total words, %d unique words\n",nwords,nunique);
		printf("Time to process on %d procs = %g (secs), %d, %d\n", nprocs, tstop-tstart, mapitem, reduceitem);
	}

	MPI_Finalize();
}


int getfilename(char *fullpath, char *filename)
{
	size_t found;
	std::string path(fullpath);
	found=path.find_last_of("/\\");
	const char *name = path.substr(found+1).c_str();
	memcpy(filename, name, strlen(name)+1);

	return strlen(name);
}

/* ----------------------------------------------------------------------
   read a file
   for each word in file, emit key = word, value = NULL
------------------------------------------------------------------------- */
void mymap(int nmap, KeyValue *kv, void *ptr)
{
	int me = *(int*)ptr;
	hipSetDevice(0);

	struct timeval	start_map, end_map;
	double time_map = 0.0;

	hipDeviceSynchronize();
	gettimeofday(&start_map, NULL);

	int resultlen;
	char hostname[20];
	MPI_Get_processor_name(hostname, &resultlen);
	int host_id = -1;
	
	if(strcmp(hostname, "master\0")==0)
		host_id = 0;
	else
	{
		sscanf(hostname, "node%d", &host_id);
		//host_id -= 1;	
	}

	int file_each_proc = num_file/nprocs;

	for(int fid=me*file_each_proc; fid<(me+1)*file_each_proc && fid < num_file; fid++)
	{
		char fullname[100];
		sprintf(fullname, "%s/part-%05d\0", inputdir, fid);

		printf("full file name and gpu id is %s, %d\n", fullname, me%4);

		// filesize = # of bytes in file
		struct stat stbuf;
		int flag = stat(fullname,&stbuf);
		if (flag < 0) {
			printf("ERROR: Could not query file size %d, %s\n", me, fullname);
			MPI_Abort(MPI_COMM_WORLD,1);
		}
		int filesize = stbuf.st_size;

		FILE *fp = fopen(fullname,"r");
		char *text = new char[filesize+1];
		int nchar = fread(text,1,filesize,fp);
		text[nchar] = '\0';
		fclose(fp);

		char filename[100];
		int namelen = getfilename(fullname, filename);

		//copy text data into gpu memory
		char *d_text;
		hipMalloc((void**)&d_text, (filesize+1)*sizeof(char));
		hipMemcpy(d_text, text, (filesize+1)*sizeof(char), hipMemcpyHostToDevice);

		//record the start position of each url
		int *d_sequence;
		int *d_segmask;

		hipMalloc((void**)&d_sequence, (filesize+1)*sizeof(int));
		hipMalloc((void**)&d_segmask, (filesize+1)*sizeof(int));

		thrust::device_ptr<int> dev_sequence(d_sequence);
		thrust::device_ptr<int> dev_segmask(d_segmask);

		thrust::sequence(dev_sequence, dev_sequence+(filesize+1));

		dim3 h_dimBlock(256,1,1);
		dim3 h_dimGrid(1,1,1);
		int numBlocks = CEIL(filesize+1, h_dimBlock.x);
		THREAD_CONF(h_dimGrid, h_dimBlock, numBlocks, h_dimBlock.x);

		//hipEvent_t start, stop;
		//float time1;
		//hipEventCreate(&start);
		//hipEventCreate(&stop);
		//hipEventRecord(start, 0);

		//record the position array (about 4ms for 64M)
		mark<<<h_dimGrid, h_dimBlock>>>(d_text, d_segmask, (filesize+1));

		//hipEventRecord(stop, 0);
		//hipEventSynchronize(stop);
		//hipEventElapsedTime(&time1, start, stop);
		//printf("time is %f\n", time1);

		//printf("zhao2 %d\n", me);

		int urlcount =	thrust::count(dev_segmask, dev_segmask+(filesize+1), 1);

		if(urlcount == 0)
			return;

		int *d_urloffset;
		int *d_urllength;

		hipMalloc((void**)&d_urloffset, urlcount*sizeof(int));
		hipMalloc((void**)&d_urllength, urlcount*sizeof(int));

		thrust::device_ptr<int> dev_urloffset(d_urloffset);

		//about 14ms
		thrust::copy_if(dev_sequence, dev_sequence+(filesize+1),
				dev_segmask, dev_urloffset, is_start());

		dim3 h_dimGrid2(1,1,1);
		dim3 h_dimBlock2(256,1,1);
		numBlocks = CEIL(urlcount, h_dimBlock2.x);
		THREAD_CONF(h_dimGrid2, h_dimBlock2, numBlocks, h_dimBlock2.x);

		//about 8ms
		compute_url_length<<<h_dimGrid2, h_dimBlock2>>>(
				d_text,
				d_urloffset,
				d_urllength,
				(filesize+1),
				urlcount);

		int *h_urloffset = new int[urlcount];
		int *h_urllength = new int[urlcount];

		hipMemcpy(text, d_text, (filesize+1)*sizeof(char), hipMemcpyDeviceToHost);
		hipMemcpy(h_urloffset, d_urloffset, urlcount*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(h_urllength, d_urllength, urlcount*sizeof(int), hipMemcpyDeviceToHost);

		//about 18ms for 64m ii data
		for(int i=0; i<urlcount; i++)
		{
			kv->add(text+h_urloffset[i], h_urllength[i]+1, filename, namelen+1);
		}

		//free device memory
		hipFree(d_text);
		hipFree(d_sequence);
		hipFree(d_segmask);
		hipFree(d_urloffset);
		hipFree(d_urllength);

		delete [] text;
		delete [] h_urloffset;
		delete [] h_urllength;
	}
	
	//printf("end of map %d\n", me);

	hipDeviceSynchronize();
	gettimeofday(&end_map, NULL);
	time_map += (1000*(end_map.tv_sec-start_map.tv_sec)
			+(end_map.tv_usec-start_map.tv_usec + 0.0)/1000);
	printf("time of %d is %f\n", me, time_map);

}


void mycombine(char *key, int keybytes, char *multivalue,
		int nvalues, int *valuebytes, KeyValue *kv, void *ptr)
{

	stringstream ss (stringstream::in | stringstream::out);

	int t = 0;
	if(nvalues)
	{
		char* curval = multivalue;
		for(int i=0; i<nvalues; i++)
		{
			if(t!=0)
				ss << " ";
			ss << curval;
			curval += valuebytes[i];
			t++;
		}
	}
	else
	{
		MapReduce *mr = (MapReduce *) valuebytes;
		int nblocks;
		uint64_t nvalues_total = mr->multivalue_blocks(nblocks);
		for (int iblock = 0; iblock < nblocks; iblock++)
		{
			int nv = mr->multivalue_block(iblock,&multivalue,&valuebytes);

			char* curval = multivalue;
			for (int i = 0; i < nv; i++)
			{
				if(t!=0)
					ss << " ";
				ss << curval;
				curval += valuebytes[i];
				t++;
				//process each value within the block of values
			}
		}
	}

	string s = ss.str();
	kv->add(key, keybytes, (char*)s.c_str(), (int)(s.length()+1));
}

/* ----------------------------------------------------------------------
   count word occurrence
   emit key = word, value = # of multi-values
------------------------------------------------------------------------- */

void myreduce(char *key, int keybytes, char *multivalue,
		int nvalues, int *valuebytes, KeyValue *kv, void *ptr)
{

	Info *info = (Info*) ptr;
	int me = info->me;
	int nproc = info->nproc;

	char filename[50];
	sprintf(filename, "/mnt/mrmpi_output/InvertedIndex-%d-%d\0", nproc , me);

	//printf("filename is %s, %d\n", filename, nvalues);

	std::fstream filestr;
	filestr.open (filename, fstream::out | fstream::app);

	filestr << key << "\t";

	if(nvalues)
	{
		char* curval = multivalue;
		for(int i=0; i<nvalues; i++)
		{
			filestr <<  curval << " ";
			curval += valuebytes[i];
		}
		filestr << endl;
	}
	else
	{
		MapReduce *mr = (MapReduce *) valuebytes;
		int nblocks;
		uint64_t nvalues_total = mr->multivalue_blocks(nblocks);
		for (int iblock = 0; iblock < nblocks; iblock++)
		{
			int nv = mr->multivalue_block(iblock,&multivalue,&valuebytes);

			char* curval = multivalue;
			for (int i = 0; i < nv; i++)
			{
				filestr <<  curval << " ";
				curval += valuebytes[i];
				//process each value within the block of values
			}
		}

		filestr << endl;
	}

	filestr.close();
}

/* ----------------------------------------------------------------------
   compare two counts
   order values by count, largest first
------------------------------------------------------------------------- */

int ncompare(char *p1, int len1, char *p2, int len2)
{
	int i1 = *(int *) p1;
	int i2 = *(int *) p2;
	if (i1 > i2) return -1;
	else if (i1 < i2) return 1;
	else return 0;
}

/* ----------------------------------------------------------------------
   process a word and its count
   depending on flag, emit KV or print it, up to limit
------------------------------------------------------------------------- */

void output(uint64_t itask, char *key, int keybytes, char *value,
		int valuebytes, KeyValue *kv, void *ptr)
{
	/*
	Count *count = (Count *) ptr;
	count->n++;
	if (count->n > count->limit) return;

	int n = *(int *) value;
	if (count->flag)
		printf("%d %s\n",n,key);
	else
		kv->add(key,keybytes,(char *) &n,sizeof(int));
	*/
}

